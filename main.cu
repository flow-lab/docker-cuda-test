#include "hip/hip_runtime.h"

#include <iostream>

// Just for testing
// code from https://stackoverflow.com/questions/13320321/printf-in-my-cuda-kernel-doesnt-result-produce-any-output

__global__
void set1(int *t) {
    t[threadIdx.x] = 1;
}

inline bool failed(hipError_t error) {
    if (hipSuccess == error)
        return false;

    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    return true;
}

int main() {
    int blockSize;
    for (blockSize = 1; blockSize < 1 << 12; blockSize++) {
        printf("Testing block size of %d\n", blockSize);
        int *t;
        if (failed(hipMallocManaged(&t, blockSize * sizeof(int)))) {
            failed(hipFree(t));
            break;
        }
        for (int i = 0; i < blockSize; i++)
            t[0] = 0;
        set1 <<<1, blockSize>>>(t);
        if (failed(hipPeekAtLastError())) {
            failed(hipFree(t));
            break;
        }
        if (failed(hipDeviceSynchronize())) {
            failed(hipFree(t));
            break;
        }

        bool hasError = false;
        for (int i = 0; i < blockSize; i++)
            if (1 != t[i]) {
                printf("CUDA error: t[%d] = %d but not 1\n", i, t[i]);
                hasError = true;
                break;
            }
        if (hasError) {
            failed(hipFree(t));
            break;
        }

        failed(hipFree(t));
    }
    blockSize--;
    if (blockSize <= 0) {
        printf("CUDA error: block size cannot be 0\n");
        return 1;
    }
    printf("Block maximum size is %d", blockSize);
    return 0;
}